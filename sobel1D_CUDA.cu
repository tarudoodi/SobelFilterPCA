#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sobel1D.cu
 Author      : Taru Doodi
 Version     : v1
 Contact	 : tarudoodi@ufl.edu / tarudoodi@gmail.com
 Copyright   : Your copyright notice
 Description : CUDA code for GPU K20x, specifically coded for [3x3] gradient masks.
 				runs for image sizes 2x2 to 2048x2048 and stores the timing data in .csv file.
 ============================================================================
 */

#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void sobel(int *d_input, int *d_output, long int N)
{
	//__shared__ int smBytes[];
	int idx = threadIdx.x;
	int bidx = blockIdx.x;
	float  sobel_x,sobel_y;
	int pixel = bidx*blockDim.x  + idx;
	if(N<=pixel && pixel<N*(N-1))
	{
		// the gradient in x direction
		sobel_x = (float) (1*(d_input[pixel -1 -N]) + 2*(d_input[pixel-N]) +1*(d_input[pixel-N+1]) - 1*(d_input[pixel +N-1]) - 2*(d_input[pixel+N]) - 1*(d_input[pixel+N+1]));
		// the gradient in y direction
		sobel_y = (float) (1*(d_input[pixel -1 -N]) - 1*(d_input[pixel -N+1]) + 2*(d_input[pixel-1]) - 2*(d_input[pixel +1]) + 1*(d_input[pixel -1 +N]) - 1*(d_input[pixel +1 +N]));
		/*
	// the gradient in x direction
	sobel_x = (double)( 1*(d_input[(idx-1)*N + idy-1]) +2*(d_input[idy-1 +idx*N]) +1*(d_input[idy-1 +N*(idx+1)]) - 1*(d_input[idy+1+N*(idx-1)]) - 2*(d_input[idy+1 + N*idx]) - 1*(d_input[idy+1 + N*(idx+1)]));
	// the gradient in y direction
	sobel_y = (double) (1*(d_input[idy-1 + N*(idx-1)]) - 1*(d_input[idy-1 +N*(idx+1)]) + 2*(d_input[idy + N*(idx-1)]) - 2*(d_input[idy +N*(idx+1)]) + 1*(d_input[idy+1 +N*(idx-1)]) - 1*(d_input[idy+1 + N*(idx+1)]));
		 */

	d_output[pixel] = (int) sqrt((sobel_x*sobel_x) + (sobel_y*sobel_y));
//d_output[pixel] =(int)  hypot(sobel_x, sobel_y);//	doesnt make any difference
if (d_output[pixel] > 255)
		{
			(d_output[pixel]) = 255;
		}
	}
    else
    {
        	d_output[pixel] = d_input[pixel];
    }
}


/* Initializing the image matrix */
void initializeImage(int *d_input,int N)
{
	for(int i=0; i<N*N; i++)
	{
		d_input[i] = (int) (rand()%255);
		//printf("%d the value of input array is %d \n", (i),h_input[i]);
	}
}

double timerval ()
{
	struct timeval st;
	gettimeofday (&st, NULL);
	return st. tv_sec + st. tv_usec * 1e-6;
}

int main()
{
	int i; // loop counters
	int m=0;
	int N=8; // rows, columns of image
	int *h_input,*h_output; //*h_x,*h_y;
	//double start_time, end_time;

	//CUDA Variables

	
	int *d_output,*d_input;
	int blockNum, threadNum;
	hipEvent_t start,end;
	hipEventCreate(&start); //timers
	hipEventCreate(&end);
	float time;

	int kStart =2;
	int kStop =12;
	double seconds[kStop];
	int k =0;
//	k = kStop;
		for(k=kStart;k<=kStop; k++)
		{
			N = pow(2,k);
			h_input = (int*)malloc(N*N*sizeof(int));
			h_output = (int*)malloc(N*N*sizeof(int));
			// allocate GPU memory
			hipMalloc((void**) &d_input, (N)*(N)*sizeof(int));
			hipMalloc((void**) &d_output,(N)*(N)*sizeof(int));
			// generate the input array on the host
			//calculate thread number and block number
			if(N<=4)//8)//32)
			{
				blockNum = 1;
				threadNum =N*N;//128;//256;//512;// N*N;
			}
			else
			{
				blockNum = N*N/32;//64;//128;//256;//512;
				threadNum = 32; //64;//128;//256;//512;
			}
			initializeImage(h_input,N);
			hipMemcpy(d_input, h_input, N*N*sizeof(int), hipMemcpyHostToDevice); // transfer the array to the GPU
			hipDeviceSynchronize();
			hipEventSynchronize(start);
			//start_time = timerval();
			hipEventRecord(start,0);
			// launch the kernel
			for(i=0;i<1000;i++)
			{
//				printf("ran for ith = %d for size k =%d",i,k);
	//			sobel_edges(d_output, d_input, N); //just do the edges of the image on CPU as they
				//sobel<<<blockNum,(threadNum+2*N)*sizeof(int) >>>(d_input, d_output,N);
				sobel<<<blockNum,threadNum>>>(d_input, d_output,N);
				hipDeviceSynchronize();
			}
			hipEventSynchronize(end);
			//end_time = timerval();
			hipEventRecord(end,0);
			hipMemcpy(h_output, d_output, N*N*sizeof(int), hipMemcpyDeviceToHost);// copy back the result array to the CPU
			hipEventElapsedTime(&time,start,end);
			//seconds[m++] = (start_time - end_time)/1000;
			seconds[m++] = time/1000;
			hipFree(d_input);
			hipFree(d_output);
			free(h_input);
			free(h_output);
		}

		// printing to file
		FILE *sobelParallelOutputfile;
		sobelParallelOutputfile = fopen("sobelParallelOutputfile.csv","a+");
		if(sobelParallelOutputfile == NULL)
		{
			printf("Could not open file\n");
			return EXIT_FAILURE;
		}
		fprintf(sobelParallelOutputfile,"N, Time taken \n");
		for(i=0;i<m;i++)
		{
			fprintf(sobelParallelOutputfile,"%lf,%f\n", pow(2,(i+kStart)), seconds[i]);
		}
		fclose(sobelParallelOutputfile); //Closing the file
		m=0; //reset m;
	return EXIT_SUCCESS;
}
